#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <errno.h>
#include <random>
#include <vector>
#include <ctime>
#include <math.h>
#include <pthread.h>
// CUDA imports
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
// Library imports
#include <curl/curl.h>
#include "json.hpp"
#include "Timer.h"

using namespace std;
using json = nlohmann::json;

/////////////////////////// Classes: ///////////////////////////
class Managed {
public:
	void *operator new(size_t len) {
		void *ptr;
		hipMallocManaged(&ptr, len);
//		hipDeviceSynchronize();
		return ptr;
	}

	void operator delete(void *ptr) {
//		hipDeviceSynchronize();
		hipFree(ptr);
	}
};

class Order : public Managed {
public:
	double price;
	double quantity;
	
	__host__ __device__ Order() {
		price = 0;
		quantity = 0;
	}
	
	__host__ __device__ Order(double p, double q) {
		price = p;
		quantity = q;
	}

	bool operator < (const Order& ord) const {
		return (price < ord.price);
	}

	bool operator > (const Order& ord) const {
		return (price > ord.price);
	}
};

class MarketBook : public Managed {
public:
	int size = 100;
	Order asks[100];
	Order bids[100];

	MarketBook() {
		hipMallocManaged((void**)&asks, size*sizeof(Order));
		hipMallocManaged((void**)&bids, size*sizeof(Order));
	}
	__device__ MarketBook(int a) {
	}
	
	__host__ __device__ double calculate_mean() {
		double total_quantity = 0;
		double total = 0;
		for(int i = 0; i != size; i++) {
			total_quantity += asks[i].quantity;
			total += asks[i].quantity*asks[i].price;
		}
		for(int i = 0; i != size; i++) {
			total_quantity += bids[i].quantity;
			total += bids[i].quantity*bids[i].price;
		}
		return total/total_quantity;
	}


	__host__ __device__ double calculate_total() {
		double total_quantity = 0;
		for(int i = 0; i != size; i++) {
			total_quantity += asks[i].quantity;
		}
		for(int i = 0; i != size; i++) {
			total_quantity += bids[i].quantity;
		}
		return total_quantity;
	}
	
	__host__ __device__ double calculate_stddev() {
		double total = calculate_total();
		double sum = 0;
		double probability = 0;

		for(int i = 0; i != size; i++) {
			probability = asks[i].quantity/total;
			sum += asks[i].price*asks[i].price*probability;
		}
		for(int i = 0; i != size; i++) {
			probability = bids[i].quantity/total;
			sum += bids[i].price*bids[i].price*probability;
		}
		double mean = calculate_mean();
		return pow((sum - mean*mean), 0.5);
	}
	
	__host__ __device__ int find_ask(double price) {
		for(int i=0; i < size-1; i++) {
			if(asks[i].price <= price) {
				if(asks[i+1].price > price) {
					return i;
				} 
			}
		}
		return -1;
	}

	__host__ __device__ int find_bid(double price) {
		for(int i=0; i < size-1; i++) {
			if(bids[i].price >= price) {
				if(bids[i+1].price < price) {
					return i;
				} 
			}
		}
		return -1;
	}
	
	__host__ __device__ int index_ask(double price) {
		for(int i=0; i < size-1; i++) {
			if(asks[i].price == price) return i;
		}
		return -1;
	}
	__host__ __device__ int index_bid(double price) {
		for(int i=0; i < size-1; i++) {
			if(bids[i].price == price) return i;
		}
		return -1;
	}	

	__host__ __device__ bool remove_ask(double price) {
		int pos = find_ask(price);
		if(pos == -1) return false;
		for(int i=pos; i < size-1; i++) {
			asks[i].price = asks[i+1].price;
			asks[i].quantity = asks[i+1].quantity;
		}
		return true;	
	}

	__host__ __device__ bool remove_bid(double price) {
		int pos = find_bid(price);
		if(pos == -1) return false;
		for(int i=pos; i < size-1; i++) {
			bids[i].price = bids[i+1].price;
			bids[i].quantity = bids[i+1].quantity;
		}
		return true;	
	}

	__host__ __device__ bool insert_ask(Order ask) {
		int i = 0; 
		for(int j=0; j<size-1; j++) {
			if(asks[j].price != 0 && asks[j+1].price ==0) {
				i = j;
			}
		}
		while ((i > 0) && (ask.price < asks[i-1].price)) {   
			asks[i].price = asks[i-1].price;
			asks[i].quantity = asks[i-1].quantity;
			i = i - 1;
		}
		asks[i].price = ask.price;
		asks[i].quantity = ask.quantity;
		return true;
	}

	__host__ __device__ bool insert_bid(Order bid) {
		int i = 0; 
		for(int j=0; j<size-1; j++) {
			if(bids[j].price != 0 && bids[j+1].price ==0) {
				i = j;
			}
		}
		while ((i > 0) && (bid.price < bids[i-1].price)) {   
			bids[i].price = bids[i-1].price;
			bids[i].quantity = bids[i-1].quantity;
			i = i - 1;
		}
		bids[i].price = bid.price;
		bids[i].quantity = bid.quantity;
		return true;
	}



	__host__ __device__ bool insert_order(Order active_order, bool ask) {
		double remaining = 0;
		int pos = 0;
		if(ask) {
			pos = find_bid(active_order.price);
			if(pos != -1) {
				if(active_order.quantity > bids[pos].quantity) {
					active_order.quantity -= bids[pos].quantity;
					remove_bid(bids[pos].price);
					return insert_order(active_order, ask);
				}else{
					bids[pos].quantity -= active_order.quantity;
					return true;
				}
			} else {
				pos = index_ask(active_order.price);
				if(pos != -1) {
					asks[pos].quantity += active_order.quantity;
				} else {
					insert_ask(active_order);
				}
				return false;
			}
		} else {
			pos = find_ask(active_order.price);
			if(pos != -1) {
				if(active_order.quantity > asks[pos].quantity) {
					active_order.quantity -= asks[pos].quantity;
					remove_ask(asks[pos].price);
					return insert_order(active_order, ask);
				}else{
					asks[pos].quantity -= active_order.quantity;
					return true;
				}
			} else {
				pos = index_bid(active_order.price);
				if(pos != -1) {
					bids[pos].quantity += active_order.quantity;
				} else {
					insert_bid(active_order);
				} 
				return false;
			}
		}
	}
	
/*
	__host__ __device__ bool sort_books() {
		sort(asks.begin(), asks.end());
		sort(bids.begin(), bids.end());
		return true;
	}
*/
	__device__ void copy_in(MarketBook book) {
		for(int i=0; i<size; i++) {
			asks[i].price = book.asks[i].price;
			asks[i].quantity = book.asks[i].quantity;
			bids[i].price = book.bids[i].price;
			bids[i].quantity = book.bids[i].quantity;
		}
	}
};


/////////////////////////// CUDA stuff: ///////////////////////////
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void setup_kernel(hiprandState *state) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(1234, id, 0, &state[id]);
}


__global__ void simulate_market(MarketBook in_book, int generations, hiprandState *state, double *means, double *devs) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	MarketBook book = MarketBook(0);
	book.copy_in(in_book);
	hiprand_init(1234, id, 0, &state[id]);
	hiprandState localState = state[id];

	double gen_mean, gen_stddev, number;
	for(int i=0; i<generations; i++) {
		gen_mean = book.calculate_mean();
		gen_stddev = book.calculate_stddev();
		number = hiprand_log_normal_double(&localState, gen_mean, gen_stddev)-1;
		if(hiprand_uniform(&localState) > 0.5) {
			book.insert_order(Order(number, 1), false);
		} else {
			book.insert_order(Order(number, 1), true);
		}
	}
	means[id] = gen_mean;
	devs[id] = gen_stddev;
}


/////////////////////////// CPU stuff: ///////////////////////////
struct thread_data
{
	double mean;
	double stddev;
	int g;
	MarketBook book;
};

static size_t WriteCallback(void *contents, size_t size, size_t nmemb, void *userp)
{
	((std::string*)userp)->append((char*)contents, size * nmemb);
	return size * nmemb;
}

std::string get_time() {
	time_t t = time(0);
	char cstr[128];
	strftime(cstr, sizeof(cstr), "%Y-%m-%d %H:%M:%S", localtime(&t) );
	return cstr;
}

std::string get_time(int ss) {
	time_t t = time(0)+ ss;
	char cstr[128];
	strftime(cstr, sizeof(cstr), "%Y-%m-%d %H:%M:%S", localtime(&t));
	return cstr;
}

int get_ts() {
	time_t t = time(0);
	return t;
}

json get_response(string URL) {
	CURL *curl;
	CURLcode res;
	string readBuffer;
	json data;
	curl = curl_easy_init();
	if(curl) {
		curl_easy_setopt(curl, CURLOPT_URL, URL.c_str());
		curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, WriteCallback);
		curl_easy_setopt(curl, CURLOPT_WRITEDATA, &readBuffer);
		res = curl_easy_perform(curl);
		curl_easy_cleanup(curl);
		data = json::parse(readBuffer.c_str());
	}
	return data;
}

MarketBook get_market_book(json data) {
	json obj;
	double amount, price;
	MarketBook book;
	for(int i=0; i<data["bids"].size(); i++){
		obj = data["bids"][i];
		amount = atof(obj["amount"].get<string>().c_str());
		price = atof(obj["price"].get<string>().c_str());
		book.bids[i] = Order(price, amount);
	}
	for(int i=0; i<data["asks"].size(); i++){
		obj = data["asks"][i];
		amount = atof(obj["amount"].get<string>().c_str());
		price = atof(obj["price"].get<string>().c_str());
		book.asks[i] = Order(price, amount);
	}
	return book;
}

void *simulate_market_cpu(void *threadarg) {
	thread_data *my_data;
	my_data = (thread_data *) threadarg;
	MarketBook book = my_data->book;
	int g = my_data->g;
	default_random_engine generator;
	double gen_mean, gen_stddev, number;
	for(int i=0; i<g; i++) {
		gen_mean = book.calculate_mean();
		gen_stddev = book.calculate_stddev();
		normal_distribution<double> distribution(gen_mean, gen_stddev);
		number = distribution(generator);
		if(rand()%2) {
			book.insert_order(Order(number, 1), false);
		} else {
			book.insert_order(Order(number, 1), true);
		}
	}
	my_data->mean = book.calculate_mean();
	my_data->stddev = book.calculate_stddev();
}


int main(int argc, char *argv[]) {
	printf("[OUTPUT][o]:b_ts, b_mean, b_std, p_ts, p_mean, p_std, a_ts, a_mean, a_std, rte(ms), gpu, trades, generations, blocksize\n");

	if(argc != 6) {
		fprintf(stderr, "Usage: %s gpu trades generations blocksize symbol\n", argv[0]);
		fprintf(stderr, "gpu=0 for cpu\n");
		exit(1);
	}


// Get the current order book
	string baseURL = "https://api.bitfinex.com/v1";
	string type = "stats";
	string symbol = argv[5];
	string URL = baseURL + "/" + type + "/" + symbol;
	json data = get_response(URL);

	double volume24 = atof(data[0]["volume"].get<string>().c_str());
	double volume1sec = volume24*1.0/(24*60*60);

	type = "book";
	URL = baseURL + "/" + type + "/" + symbol;
	data = get_response(URL);

	int gpu = atoi(argv[1]);
	int trades = atoi(argv[2]);
	int generations = atoi(argv[3]);
	int blocksize = atoi(argv[4]);
	MarketBook book = get_market_book(data);

	
	printf("[%s][r] mean: %f, std-dev: %f\n", get_time().c_str(),  book.calculate_mean(), book.calculate_stddev());

	if(gpu == 0) {
	// CPU
		double b_mean = book.calculate_mean();
		double b_stddev = book.calculate_stddev();
		int b_ts = get_ts();
		ggc::Timer t("generations");

		pthread_t threads[generations];
		thread_data thread_data_array[generations];
		pthread_attr_t th_attr;
		pthread_attr_init(&th_attr);
		pthread_attr_setdetachstate(&th_attr, PTHREAD_CREATE_JOINABLE);

		t.start();
		for(int i=0; i<generations; i++) {
			thread_data_array[i].g = generations;
			thread_data_array[i].book = book;
			pthread_create(&threads[i], &th_attr, simulate_market_cpu, (void *) &thread_data_array[i]);
		}
		for(int i=0; i<generations; i++) {
			pthread_join(threads[i], NULL);
		}
		t.stop();

		double avg_stddev = 0;
		double avg_mean = 0;
		for(int i=0; i<generations; i++) {
			avg_stddev += thread_data_array[i].stddev;
			avg_mean += thread_data_array[i].mean;
		}
		avg_stddev = avg_stddev/generations;
		avg_mean = avg_mean/generations;
		int p_ts = get_ts();
		printf("[%s][p] avg-mean: %f, avg-std-dev: %f\n", get_time().c_str(), avg_mean, avg_stddev);
		printf("[%s][i] runtime: %llu ms\n", get_time().c_str(), t.duration()/1000000);
		printf("[%s][i] price-expected-at: %s (%fs)\n", get_time().c_str(), get_time(trades*1.0/volume1sec).c_str(), trades*1.0/volume1sec);

		printf("[%s][i] sleeping for %fs\n", get_time().c_str(), trades*1.0/volume1sec);
		sleep(trades*1.0/volume1sec);
		MarketBook new_book = get_market_book(get_response(URL));
		int a_ts = get_ts();
		printf("[%s][r] mean: %f, std-dev: %f\n", get_time().c_str(), new_book.calculate_mean(), new_book.calculate_stddev());
		printf("[OUTPUT][o]:%d, %f, %f, %d, %f, %f, %d, %f, %f, %f, %d, %d, %d, %d\n", b_ts, b_mean, b_stddev, p_ts, avg_mean, avg_stddev, a_ts, new_book.calculate_mean(), new_book.calculate_stddev(), t.duration()/1000000.0, gpu, trades, generations, blocksize);
		

	} else {
		double b_mean = book.calculate_mean();
		double b_stddev = book.calculate_stddev();
		int b_ts = get_ts();
	// GPU
	// Setup
	// Create GPU timers
		hipEvent_t start, stop;
		float total;
		hipEventCreate(&start);
		hipEventCreate(&stop);

	// Get GPU
		int devID;
		devID = gpuGetMaxGflopsDeviceId();
		checkCudaErrors(hipSetDevice(devID));
		
		hiprandState *devStates;
		hiprandGenerator_t gen;
		hipMalloc((void **)&devStates, 64*64*sizeof(hiprandState));
		hipMallocManaged((void **)&book, sizeof(book));
		
		double * means;
		double * devs;
		hipMallocManaged(&means, generations*sizeof(double));
		hipMallocManaged(&devs, generations*sizeof(double));


		setup_kernel<<<ceil(1.0*generations/blocksize), blocksize>>>(devStates);

		hipEventRecord(start);

		simulate_market<<<ceil(1.0*generations/blocksize), blocksize>>>(book, trades, devStates, means, devs);

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&total, start, stop);

//		hipDeviceSynchronize();
		
		
		double avg_stddev = 0;
		double avg_mean = 0;
		for(int i=0; i<generations; i++) {
//			printf("[%d] mean: %f, std-dev: %f\n", i, means[i], devs[i]);
			avg_stddev += devs[i];
			avg_mean += means[i];
		}
		int p_ts = get_ts();
		avg_stddev = avg_stddev/generations;
		avg_mean = avg_mean/generations;
		
		printf("[%s][p] avg-mean: %f, avg-std-dev: %f\n", get_time().c_str(), avg_mean, avg_stddev);
		printf("[%s][i] runtime: %f ms\n", get_time().c_str(), total);
		printf("[%s][i] price-expected-at: %s (%fs)\n", get_time().c_str(), get_time(trades*1.0/volume1sec).c_str(), trades*1.0/volume1sec);

		printf("[%s][i] sleeping for %fs\n", get_time().c_str(), trades*1.0/volume1sec);

		sleep(trades*1.0/volume1sec);
		int a_ts = get_ts();
		MarketBook new_book = get_market_book(get_response(URL));
		printf("[%s][r] mean: %f, std-dev: %f\n", get_time().c_str(), new_book.calculate_mean(), new_book.calculate_stddev());
	
		printf("[OUTPUT][o]:%d, %f, %f, %d, %f, %f, %d, %f, %f, %f, %d, %d, %d, %d\n", b_ts, b_mean, b_stddev, p_ts, avg_mean, avg_stddev, a_ts, new_book.calculate_mean(), new_book.calculate_stddev(), total, gpu, trades, generations, blocksize);
		
	
	// CUDA cleanup
		gpuErrchk(hipEventDestroy(start));
		gpuErrchk(hipEventDestroy(stop));
	}

	return 0;
}
