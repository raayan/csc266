#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <errno.h>
#include <random>
#include <vector>
#include <math.h>
// CUDA imports
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
// Library imports
#include <curl/curl.h>
#include "json.hpp"
#include "Timer.h"

using namespace std;
using json = nlohmann::json;

/////////////////////////// Classes: ///////////////////////////
class Managed {
public:
	void *operator new(size_t len) {
		void *ptr;
		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();
		return ptr;
	}

	void operator delete(void *ptr) {
		hipDeviceSynchronize();
		hipFree(ptr);
	}
};

class Order : public Managed {
public:
	double price;
	double quantity;
	
	Order(double p, double q) {
		price = p;
		quantity = q;
	}

	bool operator < (const Order& ord) const {
		return (price < ord.price);
	}

	bool operator > (const Order& ord) const {
		return (price > ord.price);
	}
};

class MarketBook : public Managed {
public:
	double mean;
	double stdd;
	vector<Order> asks;
	vector<Order> bids;

	MarketBook() {

	}
	
	double calculate_mean() {
		double total_quantity = 0;
		double total = 0;
		for(std::vector<Order>::size_type i = 0; i != asks.size(); i++) {
			total_quantity += asks[i].quantity;
			total += asks[i].quantity*asks[i].price;
		}
		for(std::vector<Order>::size_type i = 0; i != bids.size(); i++) {
			total_quantity += bids[i].quantity;
			total += bids[i].quantity*bids[i].price;
		}
		return total/total_quantity;
	}

	double calculate_total() {
		double total = 0;
		for(std::vector<Order>::size_type i = 0; i != asks.size(); i++) {
			total += asks[i].quantity;
		}
		for(std::vector<Order>::size_type i = 0; i != bids.size(); i++) {
			total += bids[i].quantity;
		}
		return total;
	}
	
	double calculate_stddev() {
		double total = calculate_total();
		double sum = 0;
		double probability = 0;
		for(std::vector<Order>::size_type i = 0; i != asks.size(); i++) {
			probability = asks[i].quantity/total;
			sum += asks[i].price*asks[i].price*probability;
		}
		for(std::vector<Order>::size_type i = 0; i != bids.size(); i++) {
			probability = bids[i].quantity/total;
			sum += bids[i].price*bids[i].price*probability;
		}
		double mean = calculate_mean();
		return pow((sum - mean*mean), 0.5);
	}
	
	// ask=0 for bid ask=1 for ask
	bool insert_order(Order active_order, bool ask) {
		double remaining = 0;
		int pos = 0;
		if(ask) {
			pos = (lower_bound(bids.begin(), bids.end(), active_order) - bids.begin());
			if(pos < bids.size()) {
				if(bids[pos].price >= active_order.price && active_order.quantity > bids[pos].quantity) {
					active_order.quantity -= bids[pos].quantity;
					bids.erase(bids.begin()+pos);
					return insert_order(active_order, ask);
				}else{
					bids[pos].quantity -= active_order.quantity;
					return true;
				}
			} else {
				pos = (lower_bound(asks.begin(), asks.end(), active_order) - asks.begin());
				if(asks[pos].price == active_order.price) {
					asks[pos].quantity += active_order.quantity;
				} else {
					asks.insert(asks.begin()+pos, active_order);
				}
				return false;
			}
		} else {
			pos = (lower_bound(asks.begin(), asks.end(), active_order) - asks.begin());
			if(asks[pos].price <= active_order.price && pos < asks.size()) {
				if(active_order.quantity > asks[pos].quantity) {
					active_order.quantity -= asks[pos].quantity;
					asks.erase(asks.begin()+pos);
					return insert_order(active_order, ask);
				}else{
					asks[pos].quantity -= active_order.quantity;
					return true;
				}
			} else {
				pos = (lower_bound(bids.begin(), bids.end(), active_order) - bids.begin());
				if(bids[pos].price == active_order.price) {
					bids[pos].quantity += active_order.quantity;
				} else {
					bids.insert(bids.begin()+pos, active_order);
				} 
				return false;
			}
		}
	}
};

std::ostream& operator<< (std::ostream & out, Order const& data) {
	out << "Order(";
	out << "price=";
	out << data.price;
	out << ", ";
	out << "quantity=";
	out << data.quantity;
	out << ")";
	return out ;
}
std::ostream& operator<< (std::ostream & out, MarketBook const& data) {
	out << "MarketBook(";
	out << "asks=[\n";
	for(std::vector<Order>::size_type i = 0; i != data.asks.size(); i++) {
		out << data.asks[i];
		if(i < data.asks.size()-1) out << "\n";
	}
	out << "]\n";
	out << "bids=[\n";
	for(std::vector<Order>::size_type i = 0; i != data.bids.size(); i++) {
		out << data.bids[i];
		if(i < data.bids.size()-1) out << "\n";
	}
	out << "])";
	return out ;
}

/////////////////////////// CUDA stuff: ///////////////////////////
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ int convolve() {
	return 1;
}

__global__ void convolve_gpu(int blocks, int blocksize) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_count = blocks*blocksize;
}


/////////////////////////// CPU stuff: ///////////////////////////
static size_t WriteCallback(void *contents, size_t size, size_t nmemb, void *userp)
{
	((std::string*)userp)->append((char*)contents, size * nmemb);
	return size * nmemb;
}


int main(int argc, char *argv[]) {
	if(argc != 3) {
		fprintf(stderr, "Usage: %s gpu generations\n", argv[0]);
		fprintf(stderr, "gpu=0 for cpu\n");
		exit(1);
	}


// Get the current order book
	string baseURL = "https://api.bitfinex.com/v1";
	string type = "book";
	string symbol = "ltcbtc";
	string URL = baseURL + "/" + type + "/" + symbol;
	CURL *curl;
	CURLcode res;
	string readBuffer;
	json data;

	curl = curl_easy_init();
	if(curl) {
		curl_easy_setopt(curl, CURLOPT_URL, URL.c_str());
		curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, WriteCallback);
		curl_easy_setopt(curl, CURLOPT_WRITEDATA, &readBuffer);
		res = curl_easy_perform(curl);
		curl_easy_cleanup(curl);
		data = json::parse(readBuffer.c_str());
	}

	json obj;
	double amount, price;
	MarketBook book;

	for(int i=0; i<data["bids"].size(); i++){
		obj = data["bids"][i];
		amount = atof(obj["amount"].get<string>().c_str());
		price = atof(obj["price"].get<string>().c_str());
		book.bids.push_back(Order(price, amount));
	}
	for(int i=0; i<data["asks"].size(); i++){
		obj = data["asks"][i];
		amount = atof(obj["amount"].get<string>().c_str());
		price = atof(obj["price"].get<string>().c_str());
		book.asks.push_back(Order(price, amount));
	}

	cout << book.calculate_mean() << endl;
	cout << book.calculate_total() << endl;
	cout << book.calculate_stddev() << endl;
	sort(book.asks.begin(), book.asks.end());
	sort(book.bids.begin(), book.bids.end());
	int generations = atoi(argv[2]);

	if(atoi(argv[1]) == 0) {
	// CPU
		default_random_engine generator;
		
		for(int i=0; i<10; i++) {
			double gen_mean = book.calculate_mean();
			double gen_stddev = book.calculate_stddev();
			normal_distribution<double> distribution(gen_mean, gen_stddev);
			double number = distribution(generator);
			cout << number << endl;
		}


	} else {
	// GPU
	// Setup
		ggc::Timer t("generations");
	// Create GPU timers
		hipEvent_t start, stop;
		float total;
		hipEventCreate(&start);
		hipEventCreate(&stop);

	// Get GPU
		int devID;
		devID = gpuGetMaxGflopsDeviceId();
		checkCudaErrors(hipSetDevice(devID));

	/*
	// COPY TO GPU
	// create GPU ppm
		hipMalloc((void **) &d_img_r, size * sizeof(*d_img_r));

		// Copy data from host to device
		hipMemcpy(d_img_r, img.r, size * sizeof(*d_img_r), hipMemcpyHostToDevice);

		// Bind pointers
		hipMemcpy(&(d_img->r), &d_img_r, sizeof(d_img->r), hipMemcpyHostToDevice);

		ppm * d_out;
		int *d_out_r, *d_out_g, *d_out_b;

		gpuErrchk(hipMalloc((void**)&d_out, sizeof(ppm)));
		gpuErrchk(hipMemcpy(d_out, &out, sizeof(ppm), hipMemcpyHostToDevice));

		hipMalloc((void **) &d_out_r, size * sizeof(*d_out_r));
		hipMalloc((void **) &d_out_g, size * sizeof(*d_out_g));
		hipMalloc((void **) &d_out_b, size * sizeof(*d_out_b));

		// Copy data from host to device
		hipMemcpy(d_out_r, out.r, size * sizeof(*d_out_r), hipMemcpyHostToDevice);
		hipMemcpy(d_out_g, out.g, size * sizeof(*d_out_g), hipMemcpyHostToDevice);
		hipMemcpy(d_out_b, out.b, size * sizeof(*d_out_b), hipMemcpyHostToDevice);

		// Bind pointers
		hipMemcpy(&(d_out->r), &d_out_r, sizeof(d_out->r), hipMemcpyHostToDevice);
		hipMemcpy(&(d_out->g), &d_out_g, sizeof(d_out->g), hipMemcpyHostToDevice);
		hipMemcpy(&(d_out->b), &d_out_b, sizeof(d_out->b), hipMemcpyHostToDevice);

	// create GPU convo matrix
		int * d_n;
		int * d_cm;
		gpuErrchk(hipMalloc((void**)&d_n, 1*sizeof(int)));
		gpuErrchk(hipMemcpy(d_n, &n, 1*sizeof(int), hipMemcpyHostToDevice));
		gpuErrchk(hipMalloc(&d_cm, sizeof(int)*n*n));
		gpuErrchk(hipMemcpy(d_cm, cm, sizeof(int)*n*n, hipMemcpyHostToDevice));
	// Launch Kernels
		int blocksize = atoi(argv[4]);
		int blockcount = (img.ysize/1.0/blocksize)+1;

	// Start CPU timer and GPU timer
		t.start();
		gpuErrchk(hipEventRecord(start));

	// Actual kernel function
		convolve_gpu<<< blockcount, blocksize >>>(d_img, d_out, d_n, d_cm, blockcount, blocksize);
		getLastCudaError("Kernel execution failed (convolve_gpu).");

	// Stop CPU and GPU timer
		t.stop();
		gpuErrchk(hipEventRecord(stop));
		gpuErrchk(hipEventSynchronize(stop));

	// save time to total float
		gpuErrchk(hipEventElapsedTime(&total, start, stop));
		gpuErrchk(hipMemcpy(&out, d_out, sizeof(ppm), hipMemcpyDeviceToHost));

	// COPY BACK TO CPU
	//	int *out_r, *out_g, *out_b;
	//	out_b = (int *) malloc(size * sizeof(int));
	//	gpuErrchk(hipMemcpy(out_b, d_out_b, size * sizeof(*out_b), hipMemcpyDeviceToHost));
	//	out.b = out_b;


	// Print times
		printf("%d, %d, %llu, %f, %s, %s\n", blocksize, blockcount, t.duration(), total, argv[2], argv[1]);
	*/

	// CUDA cleanup
		gpuErrchk(hipEventDestroy(start));
		gpuErrchk(hipEventDestroy(stop));
	}

	return 0;
}
